#include "hip/hip_runtime.h"
//By: Marziyeh Nourian
#include "general_kernels.h"

__global__ void fixed_topology_kernel(  	unsigned* result_bit_vector,
											char* stream_sequences,
											#ifdef RODC_ON
											struct preprocessed_full_reference_char_sequence const * __restrict__  preprocessed_input,
											#else
											struct preprocessed_full_reference_char_sequence * preprocessed_input,
											#endif
											unsigned bit_chunks_per_state_vector, //number of integers in sv
											unsigned char_filled_ints_per_packet,
											unsigned num_packets,
											unsigned warp_efficient_stream_count,
											unsigned occupancy_efficient_stream_count,
											unsigned ref_block_count,
											unsigned batch_count,	//the number of word-chunks (32-bit) that holds the symbols of NFA s on each block
											unsigned accepting_states_count  ){

	//Registers
	unsigned ctof =							bit_chunks_per_state_vector;
	unsigned * u_stream_sequences = 	  			(unsigned *)stream_sequences;
	unsigned packet_idx = 						0;
	unsigned p_idx =						0;	
	unsigned char_filled_int_idx =					0;	
	char 	s_char =						0;	
	unsigned s_chars = 						0;	
	int s_char_index = 						0;

	//id of the input stream based on the thread and block count
	int stream_id = warp_efficient_stream_count*(blockIdx.x/ref_block_count) + (threadIdx.x/batch_count);
	if (stream_id > (occupancy_efficient_stream_count*warp_efficient_stream_count)-1 ) printf("stream-id exceed the allowed range!\n");

	extern __shared__ unsigned cfBV[];

	for(packet_idx=0; packet_idx < char_filled_ints_per_packet*num_packets; packet_idx+=char_filled_ints_per_packet){
	//set first state (i.e. non-anchored start state)
    	cfBV[threadIdx.x] = 0xFFFFFFFF;
	    cfBV[ctof + threadIdx.x] = 0xFFFFFFFF;

		for(char_filled_int_idx = packet_idx; char_filled_int_idx < packet_idx + char_filled_ints_per_packet; char_filled_int_idx += 1){

			s_chars = u_stream_sequences[(stream_id*char_filled_ints_per_packet*num_packets)+char_filled_int_idx];

			for(s_char_index = 3; s_char_index >= 0; s_char_index -= 1){

				s_char = (char)(  (s_chars>>(8*(3-s_char_index))) & 0x000000FFu   );

				topology_specific_traversal(s_char,preprocessed_input,cfBV,ctof,bit_chunks_per_state_vector,accepting_states_count,ref_block_count,batch_count);

			}

		}

		//Write to global memory
		fill_results_array(result_bit_vector, cfBV, ctof, bit_chunks_per_state_vector, accepting_states_count , packet_idx, &p_idx);

	}
}


__device__ unsigned match_check(char s_char,
								unsigned passed_chars, //is calculated by the compiler and depends on the src and dst of the TX
								unsigned tx_char_count,
								#ifdef RODC_ON
								struct preprocessed_full_reference_char_sequence const * __restrict__ preprocessed_input,
								#else
								struct preprocessed_full_reference_char_sequence * preprocessed_input,
								#endif
								unsigned ref_block_count,
								unsigned batch_count){//n is the number of 32-bit words that contain all N nfas
		unsigned mask_res  = 	0x00000000;

		int tx_char_iterator;
		for(tx_char_iterator = 0; tx_char_iterator < tx_char_count; tx_char_iterator++){

		unsigned mask  = 	0x00000000;
		unsigned offset_in_SOA = passed_chars*batch_count;

		unsigned p_chars_7 = 	preprocessed_input[(blockIdx.x%ref_block_count)].SOA_chunks_7_word[offset_in_SOA+(batch_count*tx_char_iterator)+(threadIdx.x%batch_count)];
		unsigned p_chars_6 =	preprocessed_input[(blockIdx.x%ref_block_count)].SOA_chunks_6_word[offset_in_SOA+(batch_count*tx_char_iterator)+(threadIdx.x%batch_count)];
		unsigned p_chars_5 =	preprocessed_input[(blockIdx.x%ref_block_count)].SOA_chunks_5_word[offset_in_SOA+(batch_count*tx_char_iterator)+(threadIdx.x%batch_count)];
		unsigned p_chars_4 =	preprocessed_input[(blockIdx.x%ref_block_count)].SOA_chunks_4_word[offset_in_SOA+(batch_count*tx_char_iterator)+(threadIdx.x%batch_count)];
		unsigned p_chars_3 =	preprocessed_input[(blockIdx.x%ref_block_count)].SOA_chunks_3_word[offset_in_SOA+(batch_count*tx_char_iterator)+(threadIdx.x%batch_count)];
		unsigned p_chars_2 =	preprocessed_input[(blockIdx.x%ref_block_count)].SOA_chunks_2_word[offset_in_SOA+(batch_count*tx_char_iterator)+(threadIdx.x%batch_count)];
		unsigned p_chars_1 =	preprocessed_input[(blockIdx.x%ref_block_count)].SOA_chunks_1_word[offset_in_SOA+(batch_count*tx_char_iterator)+(threadIdx.x%batch_count)];
		unsigned p_chars_0 =	preprocessed_input[(blockIdx.x%ref_block_count)].SOA_chunks_0_word[offset_in_SOA+(batch_count*tx_char_iterator)+(threadIdx.x%batch_count)];

		int j = 0;
		char p_char;

		for(j=3; j>=0; j-=1){
			p_char = (char)((p_chars_7>>(8*(3-j))) & 0x000000FFu);
			if(s_char==p_char){
				mask |= ( (0x1) << j );
			}
		}
		mask = mask<<4;
		for(j=3; j>=0; j-=1){
			p_char = (char)((p_chars_6>>(8*(3-j))) & 0x000000FFu);
			if(s_char==p_char){
				mask |= ( (0x1) << j );
			}
		}
		mask = mask<<4;
		for(j=3; j>=0; j-=1){
			p_char = (char)((p_chars_5>>(8*(3-j))) & 0x000000FFu);
			if(s_char==p_char){
				mask |= ( (0x1) << j );
			}
		}
		mask = mask<<4;
		for(j=3; j>=0; j-=1){
			p_char = (char)((p_chars_4>>(8*(3-j))) & 0x000000FFu);
			if(s_char==p_char){
				mask |= ( (0x1) << j );
			}
		}
		mask = mask<<4;
		for(j=3; j>=0; j-=1){
			p_char = (char)((p_chars_3>>(8*(3-j))) & 0x000000FFu);
			if(s_char==p_char){
				mask |= ( (0x1) << j );
			}
		}
		mask = mask<<4;
		for(j=3; j>=0; j-=1){
			p_char = (char)((p_chars_2>>(8*(3-j))) & 0x000000FFu);
			if(s_char==p_char){
				mask |= ( (0x1) << j );
			}
		}
		mask = mask<<4;
		for(j=3; j>=0; j-=1){
			p_char = (char)((p_chars_1>>(8*(3-j))) & 0x000000FFu);
			if(s_char==p_char){
				mask |= ( (0x1) << j );
			}
		}
		mask = mask<<4;
		for(j=3; j>=0; j-=1){
			p_char = (char)((p_chars_0>>(8*(3-j))) & 0x000000FFu);
			if(s_char==p_char){
				mask |= ( (0x1) << j );
			}
		}
		mask_res |= mask;
	}
	return mask_res;
}

__device__ void character_transitions_update(unsigned * cfBV, unsigned ctof,unsigned src, unsigned dst, unsigned mask,unsigned has_wildcard,unsigned has_neg,unsigned has_positive){

	unsigned current_index = (blockDim.x*src) + threadIdx.x;
	unsigned future_index = (blockDim.x*dst) + threadIdx.x;

	unsigned current = cfBV[current_index];

	if(has_wildcard){
		cfBV[ctof + future_index] |= current;
	}else if(has_positive){
		cfBV[ctof + future_index] |= mask & current;
	}else if(has_neg){
		cfBV[ctof + future_index] |= ~(mask) & current;
	}
}

__device__ void update_StateVector(unsigned * cfBV, unsigned ctof, unsigned bit_chunks_per_state_vector, unsigned accepting_states_count){
	int i=0;
	for(i=threadIdx.x + blockDim.x; i<bit_chunks_per_state_vector-((accepting_states_count)*blockDim.x); i+=blockDim.x){
		cfBV[i] = cfBV[ctof + i];
		cfBV[ctof + i] = 0;
	}//update current with future loop
	for(i=threadIdx.x + (bit_chunks_per_state_vector-((accepting_states_count)*blockDim.x)); i<bit_chunks_per_state_vector; i+=blockDim.x){
		cfBV[i] |= cfBV[ctof + i];
		cfBV[ctof + i] = 0;
	}//accumulative update of accepting states
}

__device__ void fill_results_array(unsigned * result_bit_vector, unsigned * cfBV, unsigned ctof, unsigned bit_chunks_per_state_vector, unsigned accepting_states_count, unsigned packet_idx, unsigned * p_idx){
	int i = 0;
	#ifdef STATE_VECTOR_DEBUG
	if( (packet_idx==0)){
		for(i=threadIdx.x; i<bit_chunks_per_state_vector; i+=blockDim.x){
			result_bit_vector[bit_chunks_per_state_vector*blockIdx.x + i] = cfBV[i];
			cfBV[i]=0;
		}
	}
	#else //Copy only accepting state vector portions


	unsigned offset = (*p_idx)*(accepting_states_count*gridDim.x*blockDim.x) + ( accepting_states_count*blockDim.x*blockIdx.x );
	for(i=(bit_chunks_per_state_vector - (accepting_states_count*blockDim.x) ) + threadIdx.x; i<bit_chunks_per_state_vector; i+=blockDim.x){
		result_bit_vector[ offset + i-((bit_chunks_per_state_vector - (accepting_states_count*blockDim.x) ))] = cfBV[i];
	}

	for(i=threadIdx.x; i<2*bit_chunks_per_state_vector; i+=blockDim.x){
		cfBV[i] = 0; 
	}
	(*p_idx)++;
	#endif
}


