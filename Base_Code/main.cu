#include "hip/hip_runtime.h"
#include "main.h"


struct general_config cf0;
int stream_count_from_command_line = 0;


/*function declaration*/
char * load_streaming_sequence_dividestream(char * input_sequence_filename);
char * load_streaming_sequence_multitrace();
int general_result_vector_from_device();
int general_postprocess_and_validate();
void* read_compiler_file();
static int parse_arguments(int argc, char **argv);

int general_initialize(){

	cudaCheckError( hipSetDevice(cf0.gpu_device) , __LINE__, __FILE__);

	/*reported variable initiation*/
	cf0.start=0;
	cf0.stop=0;
	cf0.preprocessing=0;
	cf0.stream_to_dev=0;
	cf0.kernel=0;
	cf0.result_from_dev=0;
	cf0.post_processing=0;
	char * out_name = strdup("final_test_general.csv");
	cf0.final_test_outfile = fopen(out_name, "a");	
	if(!(cf0.final_test_outfile)){
		printf("Final test output file not opening.  Exiting!\n");
		exit(-1);
	}
	time_t t = time(NULL);
	struct tm tm = *localtime(&t);
	sprintf(cf0.start_stamp, "%d-%d-%d %d:%d:%d\t", tm.tm_mon + 1, tm.tm_mday, tm.tm_year + 1900, tm.tm_hour, tm.tm_min, tm.tm_sec);
	cf0.start=gettime();//start preprocessing clock


	cf0.char_filled_ints_per_packet = cf0.packet_size / sizeof(unsigned);


	/*compiler's file inputs initiation*/
	cf0.nfa_size=0;
	cf0.ref_block_count=0;
	cf0.batch_count =0;
	cf0.batch_size=0;
	cf0.threads_per_block=0; //SIMDwidth
	cf0.warp_efficient_stream_count=0;
	cf0.occupancy_efficient_stream_count=0;
	cf0.accepting_states_count=0;
	cf0.SOA_chunk_count=0;

	/*compiler's file inputs reading from file*/
	/*read preprocessed data from compiler's input file*/
	cf0.fc_preprocessed_input_h = (struct preprocessed_full_reference_char_sequence *)read_compiler_file();

	
	if (stream_count_from_command_line > 0) cf0.occupancy_efficient_stream_count =stream_count_from_command_line;

	cf0.streams_count = cf0.warp_efficient_stream_count*cf0.occupancy_efficient_stream_count;
	cf0.blocks_count = cf0.occupancy_efficient_stream_count * cf0.ref_block_count;
	cf0.bit_chunks_per_state_vector = (cf0.threads_per_block*cf0.nfa_size) ;

	/*read streaming sequences characters from input file*/
#ifdef DIVIDE_TRACE
	cf0.fc_streaming_sequences_h = (char *)load_streaming_sequence_dividestream(cf0.stream_sequence_filename[0]);
#else
	cf0.fc_streaming_sequences_h = (char *)load_streaming_sequence_multitrace();
#endif
	fprintf(stderr,"\n---Initiation results---\n");

	fprintf(stderr,"Compiler's output filename = %s\n", cf0.compiler_output_filename);
	fprintf(stderr, "packet_per_kernel_launch = %d\n",cf0.packets_per_kernel_launch);
	fprintf(stderr, "packet_size = %d\n",cf0.packet_size);
	fprintf(stderr,"char_filled_ints_per_packet = %d\n", cf0.char_filled_ints_per_packet);
	fprintf(stderr,"gpu_device = %d\n", cf0.gpu_device);
	fprintf(stderr, "Application : %s \n",cf0.application);
	fprintf(stderr,"\nnfa_size = %d\n", cf0.nfa_size);
	fprintf(stderr,"ref_block_count = %d\n", cf0.ref_block_count);
	fprintf(stderr,"batch_count = %d\n", cf0.batch_count);
	fprintf(stderr,"batch_size = %d (Bytes)\n", cf0.batch_size);
	fprintf(stderr,"threads_per_block = %d\n", cf0.threads_per_block);
	fprintf(stderr,"warp_efficient_stream_count = %d\n", cf0.warp_efficient_stream_count);
	fprintf(stderr,"occupancy_efficient_stream_count = %d\n", cf0.occupancy_efficient_stream_count);
	fprintf(stderr,"accepting_states_count = %d\n", cf0.accepting_states_count);
	fprintf(stderr,"SOA_chunk_count = %d\n", cf0.SOA_chunk_count); //use to read preprocessed sequence from file
	fprintf(stderr,"streams_count = %d\n", cf0.streams_count);
	fprintf(stderr,"blocks_count = %d\n", cf0.blocks_count);
	fprintf(stderr,"bit_chunks_per_state_vector = %d\n", cf0.bit_chunks_per_state_vector);

	fprintf(stderr,"cf0.k = %d\n", cf0.k);
	fprintf(stderr,"cf0.d = %d\n", cf0.d);

	delete [] cf0.stream_sequence_filename;

//****************************************************************************************//
	return 0;
}

void* read_compiler_file(){

	FILE * input_file;
	input_file = fopen(cf0.compiler_output_filename, "r");
	if(!input_file){
		printf("\nthe input file from compiler was not successfully opened!\n");
	}
	fscanf(input_file,"%s",cf0.application);
	fscanf(input_file,"%d",&(cf0.nfa_size));
	fscanf(input_file,"%d",&(cf0.ref_block_count));
	fscanf(input_file,"%d",&(cf0.batch_count));
	fscanf(input_file,"%d",&(cf0.batch_size));
	fscanf(input_file,"%d",&(cf0.threads_per_block));
	fscanf(input_file,"%d",&(cf0.warp_efficient_stream_count));
	fscanf(input_file,"%d",&(cf0.occupancy_efficient_stream_count));
	fscanf(input_file,"%d",&(cf0.accepting_states_count));
	fscanf(input_file,"%d",&(cf0.SOA_chunk_count));

	struct preprocessed_full_reference_char_sequence * NFA_full_char_sequences = NULL;
	NFA_full_char_sequences = (struct preprocessed_full_reference_char_sequence *)calloc(cf0.ref_block_count, sizeof(struct preprocessed_full_reference_char_sequence));

	for (int i=0; i < cf0.ref_block_count; i++){
		/*array allocation and initiation*/
		char * array_7 = (char *)calloc( 4 * cf0.SOA_chunk_count +1, sizeof(char) );
		char * array_6 = (char *)calloc( 4 * cf0.SOA_chunk_count +1, sizeof(char) );
		char * array_5 = (char *)calloc( 4 * cf0.SOA_chunk_count +1, sizeof(char) );
		char * array_4 = (char *)calloc( 4 * cf0.SOA_chunk_count +1, sizeof(char) );
		char * array_3 = (char *)calloc( 4 * cf0.SOA_chunk_count +1, sizeof(char) );
		char * array_2 = (char *)calloc( 4 * cf0.SOA_chunk_count +1, sizeof(char) );
		char * array_1 = (char *)calloc( 4 * cf0.SOA_chunk_count +1, sizeof(char) );
		char * array_0 = (char *)calloc( 4 * cf0.SOA_chunk_count +1, sizeof(char) );
		/*filling char arrays*/
		char c = fgetc(input_file);//this should be \n
		for(int j=0; j < 4*cf0.SOA_chunk_count ; j++) array_7[j] = fgetc(input_file);
		c = fgetc(input_file);//this should be \n
		for(int j=0; j < 4*cf0.SOA_chunk_count ; j++) array_6[j] = fgetc(input_file);
		c = fgetc(input_file);//this should be \n
		for(int j=0; j < 4*cf0.SOA_chunk_count ; j++) array_5[j] = fgetc(input_file);
		c = fgetc(input_file);//this should be \n
		for(int j=0; j < 4*cf0.SOA_chunk_count ; j++) array_4[j] = fgetc(input_file);
		c = fgetc(input_file);//this should be \n
		for(int j=0; j < 4*cf0.SOA_chunk_count ; j++) array_3[j] = fgetc(input_file);
		c = fgetc(input_file);//this should be \n
		for(int j=0; j < 4*cf0.SOA_chunk_count ; j++) array_2[j] = fgetc(input_file);
		c = fgetc(input_file); //this should be \n
		for(int j=0; j < 4*cf0.SOA_chunk_count ; j++) array_1[j] = fgetc(input_file);
		c = fgetc(input_file);//this should be \n
		for(int j=0; j < 4*cf0.SOA_chunk_count ; j++) array_0[j] = fgetc(input_file);

		memcpy( (void *)  ( NFA_full_char_sequences[i].SOA_chunks_7_word ), (void *)array_7 , 4*cf0.SOA_chunk_count*sizeof(char)  );
		memcpy( (void *)  ( NFA_full_char_sequences[i].SOA_chunks_6_word ), (void *)array_6 , 4*cf0.SOA_chunk_count*sizeof(char)  );
		memcpy( (void *)  ( NFA_full_char_sequences[i].SOA_chunks_5_word ), (void *)array_5 , 4*cf0.SOA_chunk_count*sizeof(char)  );
		memcpy( (void *)  ( NFA_full_char_sequences[i].SOA_chunks_4_word ), (void *)array_4 , 4*cf0.SOA_chunk_count*sizeof(char)  );
		memcpy( (void *)  ( NFA_full_char_sequences[i].SOA_chunks_3_word ), (void *)array_3 , 4*cf0.SOA_chunk_count*sizeof(char)  );
		memcpy( (void *)  ( NFA_full_char_sequences[i].SOA_chunks_2_word ), (void *)array_2 , 4*cf0.SOA_chunk_count*sizeof(char)  );
		memcpy( (void *)  ( NFA_full_char_sequences[i].SOA_chunks_1_word ), (void *)array_1 , 4*cf0.SOA_chunk_count*sizeof(char)  );
		memcpy( (void *)  ( NFA_full_char_sequences[i].SOA_chunks_0_word ), (void *)array_0 , 4*cf0.SOA_chunk_count*sizeof(char)  );
		/*free the char arrays*/
		free(array_7);
		free(array_6);
		free(array_5);
		free(array_4);
		free(array_3);
		free(array_2);
		free(array_1);
		free(array_0);
	}
	fclose(input_file);
	return (void *)NFA_full_char_sequences;
}
/* reads streams into a single array from input file*/
char * load_streaming_sequence_dividestream(char * input_sequence_filename){

	char * stream_sequences = (char *)calloc(cf0.packets_per_kernel_launch * cf0.packet_size * cf0.streams_count , sizeof(char));
	FILE * infile1;
	infile1 = fopen(input_sequence_filename, "r");
	if(!infile1){
		printf("Error opening streaming sequence input file.  Check if it exists!  Exiting!\n");
		exit(-1);
	}
	unsigned result = fread (stream_sequences,1,cf0.packet_size * cf0.packets_per_kernel_launch * cf0.streams_count ,infile1);
	 if (result != cf0.packet_size*cf0.packets_per_kernel_launch* cf0.streams_count) fputs ("Not enough inputs for the streams\n",stderr);
	fclose(infile1);
	return stream_sequences;
}

char * load_streaming_sequence_multitrace(){

	FILE ** trace_file =(FILE **) malloc(cf0.trace_num*sizeof(FILE *));
	char * stream_sequences = (char *)calloc(cf0.packets_per_kernel_launch * cf0.packet_size * cf0.streams_count , sizeof(char));
	for(int i=0;i <cf0.trace_num;i++) trace_file[i] = fopen(cf0.stream_sequence_filename[i], "r");

	unsigned origin = ftell(trace_file[0]);

	for (unsigned s=0;s<cf0.streams_count;s++){

			unsigned t = s % cf0.trace_num;

			char * buffer;
			buffer = (char *) malloc (sizeof(char)*cf0.packets_per_kernel_launch * cf0.packet_size);
			fseek(trace_file[t], origin, SEEK_SET);

			unsigned result =fread(buffer, 1, cf0.packets_per_kernel_launch * cf0.packet_size ,trace_file[t]);

			if (result != cf0.packets_per_kernel_launch * cf0.packet_size) fputs ("Not enough inputs for the streams in trace file\n",stderr);
			for(int j=0;j<cf0.packets_per_kernel_launch * cf0.packet_size;j++) stream_sequences[s*(cf0.packets_per_kernel_launch*cf0.packet_size)+j]=buffer[j];

			free(buffer);
	}

	for(int i=0;i <cf0.trace_num;i++) fclose(trace_file[i]);
	delete trace_file;

	return stream_sequences;
}

int general_preprocessed_to_device(){

	cudaCheckError( hipMalloc(&(cf0.fc_preprocessed_input_d), sizeof(struct preprocessed_full_reference_char_sequence)*cf0.ref_block_count ) , __LINE__, __FILE__);
	cudaCheckError( hipMemcpy(cf0.fc_preprocessed_input_d, cf0.fc_preprocessed_input_h, sizeof(struct preprocessed_full_reference_char_sequence)*cf0.ref_block_count , hipMemcpyHostToDevice), __LINE__, __FILE__);

	cf0.stop=gettime();
	cf0.preprocessing=cf0.stop-cf0.start;

	return 0;
}

int general_stream_burst_to_device(){
	//Memory operations on/to device
	cf0.start=gettime();
	/*result bit vector allocation and initiation*/
#ifdef STATE_VECTOR_DEBUG
	cf0.result_bit_vector_h = (unsigned *)calloc(          cf0.bit_chunks_per_state_vector , sizeof(unsigned) );
	cudaCheckError( hipMalloc(&(cf0.result_bit_vector_d), cf0.bit_chunks_per_state_vector * sizeof(unsigned) ) , __LINE__, __FILE__);
	cudaCheckError( hipMemset(cf0.result_bit_vector_d, 0, cf0.bit_chunks_per_state_vector * sizeof(unsigned) ) , __LINE__, __FILE__);
#else
	cf0.result_bit_vector_h = (unsigned *)calloc(          (cf0.accepting_states_count)*cf0.blocks_count*cf0.threads_per_block*cf0.packets_per_kernel_launch,sizeof(unsigned) );
	cudaCheckError( hipMalloc(&(cf0.result_bit_vector_d), (cf0.accepting_states_count)*cf0.blocks_count*cf0.threads_per_block*cf0.packets_per_kernel_launch*sizeof(unsigned) ) , __LINE__, __FILE__);
	cudaCheckError( hipMemset(cf0.result_bit_vector_d, 0, (cf0.accepting_states_count)*cf0.blocks_count*cf0.threads_per_block*cf0.packets_per_kernel_launch*sizeof(unsigned) ) , __LINE__, __FILE__);
#endif

	cudaCheckError( hipMalloc(&(cf0.fc_streaming_sequences_d), sizeof(char)*cf0.packet_size*cf0.packets_per_kernel_launch *cf0.streams_count) , __LINE__, __FILE__);
	cudaCheckError( hipMemcpy(cf0.fc_streaming_sequences_d, cf0.fc_streaming_sequences_h, sizeof(char)*cf0.packet_size*cf0.packets_per_kernel_launch*cf0.streams_count , hipMemcpyHostToDevice), __LINE__, __FILE__);
	cf0.stop=gettime();
	cf0.stream_to_dev=cf0.stop-cf0.start;

	return 0;
}

int general_nfa_execute(){

	cf0.start = gettime();

	printf("\nKERNEL CONFIG:B: %d, T: %d.\n",cf0.blocks_count,cf0.threads_per_block);
	fixed_topology_kernel<<<cf0.blocks_count,cf0.threads_per_block,2*sizeof(unsigned)*cf0.bit_chunks_per_state_vector>>>(//2 multiplier because of requiring both current and future state vectors
																									   cf0.result_bit_vector_d,
																									   cf0.fc_streaming_sequences_d,
																									   cf0.fc_preprocessed_input_d,
																									   cf0.bit_chunks_per_state_vector,
																									   cf0.char_filled_ints_per_packet,
																									   cf0.packets_per_kernel_launch,
																									   cf0.warp_efficient_stream_count,
																									   cf0.occupancy_efficient_stream_count,
																									   cf0.ref_block_count,
																									   cf0.batch_count,
																									   cf0.accepting_states_count  );

	hipDeviceSynchronize();
	cf0.stop=gettime();
	cf0.kernel+=cf0.stop-cf0.start;


	cf0.start = gettime();

	general_result_vector_from_device();

	cf0.stop = gettime();
	cf0.result_from_dev+=cf0.stop-cf0.start;

	hipProfilerStop();

	cf0.start=gettime();

	general_postprocess_and_validate();

	cf0.stop=gettime();
	cf0.post_processing+=cf0.stop-cf0.start;

	fprintf(cf0.final_test_outfile, "%s	", cf0.start_stamp);
	fprintf(cf0.final_test_outfile, "%s	", cf0.application);
	fprintf(cf0.final_test_outfile, "blocks=%d\t", cf0.blocks_count);
	fprintf(cf0.final_test_outfile, "%3.6f\t", cf0.preprocessing);
	fprintf(cf0.final_test_outfile, "%3.6f\t", cf0.stream_to_dev);
	fprintf(cf0.final_test_outfile, "%3.6f\t", cf0.kernel);
	fprintf(cf0.final_test_outfile, "%3.6f\t", cf0.result_from_dev);
	fprintf(cf0.final_test_outfile, "%3.6f\t\n", cf0.post_processing);


	hipProfilerStart();

	return 0;
}

int general_inputstreaming(){
	
	/*move the result bit vector and the input sequence to the device*/
	general_stream_burst_to_device();

	general_nfa_execute();

	free(cf0.fc_preprocessed_input_h);
	cudaCheckError( hipFree(cf0.fc_preprocessed_input_d) , __LINE__, __FILE__);

	hipProfilerStop();


	return 0;
}


int general_result_vector_from_device(){
	//Memory operations from device
	#ifdef STATE_VECTOR_DEBUG
		cudaCheckError( hipMemcpy(cf0.result_bit_vector_h, cf0.result_bit_vector_d, cf0.bit_chunks_per_state_vector * sizeof(unsigned) , hipMemcpyDeviceToHost), __LINE__, __FILE__);
	#else
		cudaCheckError( hipMemcpy(cf0.result_bit_vector_h, cf0.result_bit_vector_d, (cf0.accepting_states_count)*cf0.blocks_count*cf0.threads_per_block*cf0.packets_per_kernel_launch*sizeof(unsigned), hipMemcpyDeviceToHost), __LINE__, __FILE__);
	#endif
	hipDeviceSynchronize();
	hipFree(cf0.result_bit_vector_d);

	free(cf0.fc_streaming_sequences_h);
	cudaCheckError( hipFree(cf0.fc_streaming_sequences_d) , __LINE__, __FILE__);


	return 0;
}

unsigned numberOfSetBits(unsigned i){
     i = i - ((i >> 1) & 0x55555555);
     i = (i & 0x33333333) + ((i >> 2) & 0x33333333);
     return (((i + (i >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
}

int general_postprocess_and_validate(){

	unsigned total_matches=0;
	unsigned line_matches=0;
#ifdef POSTPROC
	/*print the accepting portion of status vector*/
	if(!(strcmp(cf0.application,"Levenshtein")== 0) && !(strcmp(cf0.application,"Hamming")== 0)){

#ifndef STATE_VECTOR_DEBUG
		for(int i=0; i < (cf0.accepting_states_count)*cf0.threads_per_block*cf0.blocks_count*cf0.packets_per_kernel_launch; i++){
			line_matches = numberOfSetBits(cf0.result_bit_vector_h[i]);
			total_matches += line_matches;
			printf("accepting %dth -- %08x / match# : %d\n", (i%((cf0.accepting_states_count)*cf0.threads_per_block))/cf0.threads_per_block, cf0.result_bit_vector_h[i], line_matches);
		}
		printf("total_matches = %d\n",total_matches);
#else
		
		int global_id=0;
		for( int i=0;i<cf0.nfa_size; i++){
			printf("state %ds -- ",i);
			for(int j =0; j< cf0.threads_per_block;j++){
				printf("%08x ", cf0.result_bit_vector_h[global_id]);
				global_id++;
			}
			printf("\n");
		}
#endif
	}
	 if (strcmp(cf0.application,"Hamming")== 0){

#ifdef STATE_VECTOR_DEBUG
		//hamming distance validate/print
		unsigned buffer[30][30] = {0};
		int i,j,p;
		int depth = 1;
		int counter = 0;
		int w = 0;
		for(w=0; w<cf0.threads_per_block; w++){
			printf("\nThread %d's workload------\n",w);

			int loc = 0;
			for(i=0; i<cf0.k+1; i++){
				for(j=0; j<min(i+1, cf0.d+1); j++){
					for(p=0; p<cf0.threads_per_block; p++){
						if(p==w){
							buffer[i][j] = cf0.result_bit_vector_h[  (loc)*cf0.threads_per_block   +   j*cf0.threads_per_block   +   p];
						}
					}
				}
				loc+=min(i+1,cf0.d+1);
			}

			for(j=0; j<cf0.d+1; j++){
				for(i=j ; i<cf0.k+1; i++){
					printf("%08x ", buffer[i][j]);
				}
				printf("\n");
				for(p=0; p<j+1; p++){
					printf("         ");
				}
			}
			printf("\n----------()----------------\n");
		}
#else
		unsigned i = 0;
		for(i=0; i < (cf0.accepting_states_count)*cf0.blocks_count*cf0.threads_per_block*cf0.packets_per_kernel_launch; i++){
			line_matches = numberOfSetBits(cf0.result_bit_vector_h[i]);
			total_matches += line_matches;
			printf("D=%d -- %08x / match# : %d\n", i%(cf0.accepting_states_count), cf0.result_bit_vector_h[i],line_matches);
		}
		printf("total_matches = %d\n",total_matches);
	#endif
	}
	 if (strcmp(cf0.application,"Levenshtein")== 0){
#ifdef STATE_VECTOR_DEBUG

		int i,j,p;
		int w = 0;
		for(w=0; w<cf0.threads_per_block; w++){
			printf("\nThread %d's workload------\n",w);

			for(i=0; i< (cf0.d+1); i++){
				for(j=i; j< i+(cf0.k+1)*(cf0.d+1); j+=(cf0.d+1) ){
					for(p=0; p<cf0.threads_per_block; p++){
						if(p==w){	
							printf("%08x ", cf0.result_bit_vector_h[j*cf0.threads_per_block  + p]);
						}
					}
				}
				printf("\n");
			}
			printf("--------------------------\n");
		}
#else
		unsigned i = 0;
		for(i=0; i < (cf0.accepting_states_count)*cf0.threads_per_block*cf0.blocks_count*cf0.packets_per_kernel_launch; i++){
			line_matches = numberOfSetBits(cf0.result_bit_vector_h[i]);
			total_matches += line_matches;
			printf("accepting %dth -- %08x / match# : %d\n", (i%((cf0.accepting_states_count)*cf0.threads_per_block))/cf0.threads_per_block, cf0.result_bit_vector_h[i],line_matches);
		}
		printf("total_matches = %d\n",total_matches);
#endif
	}

#endif
	free(cf0.result_bit_vector_h);
	return 0;
}

/*
 *  MAIN - entry point
 */
int main(int argc, char **argv){

		parse_arguments(argc,argv);

		general_initialize();

		general_preprocessed_to_device();

		general_inputstreaming();

	return 0;
}

/* parse the main call parameters */
static int parse_arguments(int argc, char **argv)
{
	int i=1;
    if (argc < 2) {

		printf("arguments number wrong\n");
		return 0;
    }
    while(i<argc){
    	if(strcmp(argv[i], "-af") == 0 || strcmp(argv[i], "--automata_file") == 0){
    		i++;
    		if(i==argc){
    			fprintf(stderr," Automata transition file base name missing!\n");
    			return 0;
    		}
    		strcpy(cf0.compiler_output_filename ,argv[i]);
	}else if(strcmp(argv[i], "-tnum") == 0 || strcmp(argv[i], "--trace_num") == 0){
    		i++;
    		if(i==argc){
    			fprintf(stderr,"Number of trace files missing.\n");
    			return 0;
    		}
    		cf0.trace_num= atoi(argv[i]);
		i++;
		if(strcmp(argv[i], "-tnames") == 0 || strcmp(argv[i], "--tracefile_names") == 0){
			i++;
    			if(i==argc){
    				fprintf(stderr,"Name of trace files missing.\n");
    				return 0;
    			}
			cf0.stream_sequence_filename= new char *[cf0.trace_num];
			char **temp = cf0.stream_sequence_filename;
			for(int j = 0; j < cf0.trace_num; j++){ 
    				temp[j]=argv[i]; i++;
			}
			i --;
		}
		else{
			fprintf(stderr,"Name of trace files should follow number of traces.\n");
    			return 0;
		}

    	}else if(strcmp(argv[i], "-pn") == 0 || strcmp(argv[i], "--pkt_num") == 0){
    		i++;
    		if(i==argc){
    			fprintf(stderr," number of packets in each kernel call missing!\n");
    			return 0;
    		}
    		cf0.packets_per_kernel_launch= atoi(argv[i]);
    	}else if(strcmp(argv[i], "-ps") == 0 || strcmp(argv[i], "--pkt_size") == 0){
    		i++;
    		if(i==argc){
    			fprintf(stderr," size of packets missing!\n");
    			return 0;
    		}
    		cf0.packet_size=atoi(argv[i]);
    	}else if(strcmp(argv[i], "-tn") == 0 || strcmp(argv[i], "--thread_num") == 0){
    		i++;
    		if(i==argc){
    			fprintf(stderr," number of threads missing!\n");
    			return 0;
    		}
    		cf0.threads_per_block= atoi(argv[i]);
    	}else if(strcmp(argv[i], "-dev") == 0 || strcmp(argv[i], "--device") == 0){
    		i++;
    		if(i==argc){
    			fprintf(stderr," device ID missing!\n");
    			return 0;
    		}
    		cf0.gpu_device=atoi(argv[i]);
    	}else if(strcmp(argv[i], "-d") == 0 || strcmp(argv[i], "--distance") == 0){
    		i++;
    		if(i==argc){
    			fprintf(stderr," hamming or levenshtein distance missing!\n");
    			return 0;
    		}
    		cf0.d=atoi(argv[i]);
    	}else if(strcmp(argv[i], "-k") == 0 || strcmp(argv[i], "--kmer_size") == 0){
    		i++;
    		if(i==argc){
    			fprintf(stderr," size of k-mer missing!\n");
    			return 0;
    		}
    		cf0.k=atoi(argv[i]);
    	}else if(strcmp(argv[i], "--stream_count") == 0 ){
    		i++;
    		if(i==argc){
    			fprintf(stderr," cf0.occupancy_efficient_stream_count missing!\n");
    			return 0;
    		}
    		stream_count_from_command_line=atoi(argv[i]);
    	}else{
    		fprintf(stderr,"Ignoring invalid option %s\n",argv[i]);
    	}
    	i++;
    }
	return 1;
}
